#include "hip/hip_runtime.h"
/*  This program explores the order in which
    the threads operates in the CUDA. The print pattern
    suggests that there is no order.
*/
#include "stdio.h"
#include "cuPrintf.cu"
#include "cuPrintf.cuh"

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void helloCUDA()
{
    cuPrintf("Hello World!! This is thread %d.\n", blockIdx.x);
}

int main()
{
	cudaPrintfInit();
    helloCUDA<<<NUM_BLOCKS, BLOCK_WIDTH>>>();
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
    hipDeviceSynchronize();
	printf("Thats all!!!\n");
    return 0;
}
